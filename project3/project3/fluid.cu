
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <vector>
#include <cstring>
#include <stdlib.h>
#include <string>
#include <sys/time.h>

using namespace std;
//------------------------------------------------------------------------
// GPGPU Helper Routines

float *allocate_gpu(int nfloats, string name) {
    float *ret;
    hipError_t err = hipMalloc(&ret, nfloats * sizeof(float));
    if (err != hipSuccess){
        cerr << "hipMalloc for " << name << " failed" << endl;
        cerr << "error = " << hipGetErrorString(err) << endl;
        exit(-1);
    }
    return ret;
}

void free_gpu(float *ptr){
    hipError_t err = hipFree(ptr);
}

void copy_gpu_to_cpu(float *gpu, float *cpu, int allocsize, string name){
    hipError_t err = hipMemcpy(cpu, gpu, allocsize * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        cerr << "hipMemcpy for " << name << " failed" << endl;
        cerr << "error = " << hipGetErrorString(err) << endl;
        exit(-1);
    }
}

void copy_cpu_to_gpu(float *cpu, float *gpu, int allocsize, string name){
    hipError_t err = hipMemcpy(gpu, cpu, allocsize * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        cerr << "hipMemcpy for " << name << " failed" << endl;
        cerr << "error = " << hipGetErrorString(err) << endl;
        exit(-1);
    }
}

//------------------------------------------------------------------------
// Compute initial conditions for canonical Taylor-Green vortex problem
// This is an example of a CUDA kernel for the initialization routine
__global__ void setInitialConditions_kernel(float *p, float *u, float *v, float *w,
                                            int ni, int nj, int nk, int kstart,
                                            int iskip, int jskip, float L){
    const float l = 1.0;
    const float coef = 1.0;
    float dx = (1. / ni) * L;
    float dy = (1. / nj) * L;
    float dz = (1. / nk) * L;
    // The i iteration is assigned to thread blocks
    int i = blockIdx.x;
    // The k iteration is assigned to the threads within each thread block
    // (Note, this allows for coalesced memory accesses)
    int k = threadIdx.x;
    // The j iteration is performed within each thread block
    // But this could also have been added to the the thread dimensions
    for (int j = 0; j < nj; ++j){
        float x = 0.5 * dx + (i)*dx - 0.5 * L;
        float y = 0.5 * dy + j * dy - 0.5 * L;
        int offset = kstart + i * iskip + j * jskip;
        int indx = offset + k;
        float z = 0.5 * dz + k * dz - 0.5 * L;
        // 3-D taylor green vortex
        u[indx] = 1. * coef * sin(x / l) * cos(y / l) * cos(z / l);
        v[indx] = -1. * coef * cos(x / l) * sin(y / l) * cos(z / l);
        p[indx] = (1. / 16.) * coef * coef * (cos(2. * x / l) + cos(2. * y / l)) * (cos(2. * z / l) + 2.);
        w[indx] = 0;
    }
}


// Apply periodic boundary conditions at the boundary of the box
__global__ void copyPeriodic_kernel(float *p, float *u, float *v, float *w,
                int ni, int nj, int nk, int kstart, int iskip, int jskip){
    const int kskip = 1;
    // The i iteration is assigned to thread blocks
    int i = blockIdx.x;
    // The k iteration is assigned to the threads within each thread block
    int k = threadIdx.x;
    // copy the i periodic faces
    int indx = kstart + i * jskip + k * kskip;
    int offset = kstart + i * iskip;
    p[indx - iskip] = p[indx + (ni - 1) * iskip];
    p[indx - 2 * iskip] = p[indx + (ni - 2) * iskip];
    p[indx + ni * iskip] = p[indx];
    p[indx + (ni + 1) * iskip] = p[indx + iskip];

    u[indx - iskip] = u[indx + (ni - 1) * iskip];
    u[indx - 2 * iskip] = u[indx + (ni - 2) * iskip];
    u[indx + ni * iskip] = u[indx];
    u[indx + (ni + 1) * iskip] = u[indx + iskip];

    v[indx - iskip] = v[indx + (ni - 1) * iskip];
    v[indx - 2 * iskip] = v[indx + (ni - 2) * iskip];
    v[indx + ni * iskip] = v[indx];
    v[indx + (ni + 1) * iskip] = v[indx + iskip];

    w[indx - iskip] = w[indx + (ni - 1) * iskip];
    w[indx - 2 * iskip] = w[indx + (ni - 2) * iskip];
    w[indx + ni * iskip] = w[indx];
    w[indx + (ni + 1) * iskip] = w[indx + iskip];
    // copy the j periodic faces
    //for (int i = 0; i < ni; ++i){
        //int offset = kstart + i * iskip;
        //for (int k = 0; k < nk; ++k){
    indx = offset + k * kskip;
    p[indx - jskip] = p[indx + (nj - 1) * jskip];
    p[indx - 2 * jskip] = p[indx + (nj - 2) * jskip];
    p[indx + nj * jskip] = p[indx];
    p[indx + (nj + 1) * jskip] = p[indx + jskip];

    u[indx - jskip] = u[indx + (nj - 1) * jskip];
    u[indx - 2 * jskip] = u[indx + (nj - 2) * jskip];
    u[indx + nj * jskip] = u[indx];
    u[indx + (nj + 1) * jskip] = u[indx + jskip];

    v[indx - jskip] = v[indx + (nj - 1) * jskip];
    v[indx - 2 * jskip] = v[indx + (nj - 2) * jskip];
    v[indx + nj * jskip] = v[indx];
    v[indx + (nj + 1) * jskip] = v[indx + jskip];

    w[indx - jskip] = w[indx + (nj - 1) * jskip];
    w[indx - 2 * jskip] = w[indx + (nj - 2) * jskip];
    w[indx + nj * jskip] = w[indx];
    w[indx + (nj + 1) * jskip] = w[indx + jskip];

    // copy the k periodic faces
    indx = k * jskip + offset;
    p[indx - kskip] = p[indx + (nk - 1) * kskip];
    p[indx - 2 * kskip] = p[indx + (nk - 2) * kskip];
    p[indx + nk * kskip] = p[indx];
    p[indx + (nk + 1) * kskip] = p[indx + kskip];

    u[indx - kskip] = u[indx + (nk - 1) * kskip];
    u[indx - 2 * kskip] = u[indx + (nk - 2) * kskip];
    u[indx + nk * kskip] = u[indx];
    u[indx + (nk + 1) * kskip] = u[indx + kskip];

    v[indx - kskip] = v[indx + (nk - 1) * kskip];
    v[indx - 2 * kskip] = v[indx + (nk - 2) * kskip];
    v[indx + nk * kskip] = v[indx];
    v[indx + (nk + 1) * kskip] = v[indx + kskip];

    w[indx - kskip] = w[indx + (nk - 1) * kskip];
    w[indx - 2 * kskip] = w[indx + (nk - 2) * kskip];
    w[indx + nk * kskip] = w[indx];
    w[indx + (nk + 1) * kskip] = w[indx + kskip];
}


// Before summing up fluxes, zero out the residual term
__global__ void zeroResidual_kernel(float *presid, float *uresid, float *vresid, float *wresid,
                int ni, int nj, int nk, int kstart, int iskip, int jskip){
    // SERIAL PART for edge values:
    int i_edge[2] = {-1, ni};
    int k_edge[2] = {-1, nk};
    for (int i_idx = 0; i_idx < 2; i_idx++) {
        for (int k_idx = 0; k_idx < 2; k_idx++) {
            for (int j = -1; j < nj + 1; ++j){
                int offset = kstart + i_edge[i_idx]*iskip + j*jskip;
                int idx = k_edge[k_idx] + offset;
                presid[idx] = 0;
                uresid[idx] = 0;
                vresid[idx] = 0;
                wresid[idx] = 0;
            }
        }
    }
    // The i iteration is assigned to thread blocks
    int i = blockIdx.x;
    // The k iteration is assigned to the threads within each thread block
    int k = threadIdx.x;
    for (int j = -1; j < nj + 1; ++j){
        int offset = kstart + i*iskip + j*jskip;
        int idx = k + offset;
        presid[idx] = 0;
        uresid[idx] = 0;
        vresid[idx] = 0;
        wresid[idx] = 0;
    }
}

__global__ void computeResidual_x_kernel(float *presid, float *uresid, float *vresid, float *wresid,
                    const float *p, const float *u, const float *v, const float *w,
                    float eta, float nu, float dx, float dy, float dz,
                    int ni, int nj, int nk, int kstart, int iskip, int jskip) {
    const float vcoef = nu / dx;
    const float area = dy * dz;
    // i dimension goes in the +x coordinate direction
    int j = blockIdx.x;  // j iteration is assigned to thread blocks
    int k = threadIdx.x; // k iteration is assigned to the threads within each thread block
    // Loop through i faces of the mesh and compute fluxes in x direction
    // Add fluxes to cells that neighbor face
    for (int i = 0; i < ni+1; i++){
        int offset = kstart + i * iskip + j * jskip;
        const int indx = k + offset;
        // Compute the x direction inviscid flux; extract pressures from the stencil
        float ull = u[indx - 2 * iskip];
        float ul = u[indx - iskip];
        float ur = u[indx];
        float urr = u[indx + iskip];

        float vll = v[indx - 2 * iskip];
        float vl = v[indx - iskip];
        float vr = v[indx];
        float vrr = v[indx + iskip];

        float wll = w[indx - 2 * iskip];
        float wl = w[indx - iskip];
        float wr = w[indx];
        float wrr = w[indx + iskip];

        float pll = p[indx - 2 * iskip];
        float pl = p[indx - iskip];
        float pr = p[indx];
        float prr = p[indx + iskip];
        float pterm = (2. / 3.) * (pl + pr) - (1. / 12.) * (pl + pr + pll + prr);
        // x direction so the flux will be a function of u
        float udotn1 = ul + ur;
        float udotn2 = ul + urr;
        float udotn3 = ull + ur;
        float pflux = eta * ((2. / 3.) * udotn1 - (1. / 12.) * (udotn2 + udotn3));
        float uflux = ((1. / 3.) * (ul + ur) * udotn1 -
                        (1. / 24.) * ((ul + urr) * udotn2 + (ull + ur) * udotn3) + pterm);
        float vflux = ((1. / 3.) * (vl + vr) * udotn1 -
                        (1. / 24.) * ((vl + vrr) * udotn2 + (vll + vr) * udotn3));

        float wflux = ((1. / 3.) * (wl + wr) * udotn1 -
                        (1. / 24.) * ((wl + wrr) * udotn2 + (wll + wr) * udotn3));

        // Add in viscous fluxes integrate over face area
        pflux *= area;
        uflux = area * (uflux - vcoef * ((5. / 4.) * (ur - ul) - (1. / 12.) * (urr - ull)));
        vflux = area * (vflux - vcoef * ((5. / 4.) * (vr - vl) - (1. / 12.) * (vrr - vll)));
        wflux = area * (wflux - vcoef * ((5. / 4.) * (wr - wl) - (1. / 12.) * (wrr - wll)));

        presid[indx - iskip] -= pflux;
        uresid[indx - iskip] -= uflux;
        vresid[indx - iskip] -= vflux;
        wresid[indx - iskip] -= wflux;
        __syncthreads();
        presid[indx] += pflux;
        uresid[indx] += uflux;
        vresid[indx] += vflux;
        wresid[indx] += wflux;
    }
}

__global__ void computeResidual_y_kernel(float *presid, float *uresid, float *vresid, float *wresid,
                    const float *p, const float *u, const float *v, const float *w,
                    float eta, float nu, float dx, float dy, float dz,
                    int ni, int nj, int nk, int kstart, int iskip, int jskip) {
    const float vcoef = nu / dy;
    const float area = dx * dz;
    // j dimension goes in the +y coordinate direction
    int i = blockIdx.x;  // i iteration is assigned to thread blocks
    int k = threadIdx.x; // k iteration is assigned to the threads within each thread block
     // Loop through j faces of the mesh and compute fluxes in y direction
    // Add fluxes to cells that neighbor face
    for (int j = 0; j < nj+1; j++){
        int offset = kstart + i * iskip + j * jskip;
        const int indx = k + offset;
        // Compute the y direction inviscid flux
        // extract pressures and velocity from the stencil
        float ull = u[indx - 2 * jskip];
        float ul = u[indx - jskip];
        float ur = u[indx];
        float urr = u[indx + jskip];

        float vll = v[indx - 2 * jskip];
        float vl = v[indx - jskip];
        float vr = v[indx];
        float vrr = v[indx + jskip];

        float wll = w[indx - 2 * jskip];
        float wl = w[indx - jskip];
        float wr = w[indx];
        float wrr = w[indx + jskip];

        float pll = p[indx - 2 * jskip];
        float pl = p[indx - jskip];
        float pr = p[indx];
        float prr = p[indx + jskip];
        float pterm = (2. / 3.) * (pl + pr) - (1. / 12.) * (pl + pr + pll + prr);
        // y direction so the flux will be a function of v
        float udotn1 = vl + vr;
        float udotn2 = vl + vrr;
        float udotn3 = vll + vr;
        float pflux = eta * ((2. / 3.) * udotn1 - (1. / 12.) * (udotn2 + udotn3));
        float uflux = ((1. / 3.) * (ul + ur) * udotn1 -
                        (1. / 24.) * ((ul + urr) * udotn2 + (ull + ur) * udotn3));

        float vflux = ((1. / 3.) * (vl + vr) * udotn1 -
                        (1. / 24.) * ((vl + vrr) * udotn2 + (vll + vr) * udotn3) + pterm);

        float wflux = ((1. / 3.) * (wl + wr) * udotn1 -
                        (1. / 24.) * ((wl + wrr) * udotn2 + (wll + wr) * udotn3));

        // Add in viscous fluxes integrate over face area
        pflux *= area;
        uflux = area * (uflux - vcoef * ((5. / 4.) * (ur - ul) - (1. / 12.) * (urr - ull)));
        vflux = area * (vflux - vcoef * ((5. / 4.) * (vr - vl) - (1. / 12.) * (vrr - vll)));
        wflux = area * (wflux - vcoef * ((5. / 4.) * (wr - wl) - (1. / 12.) * (wrr - wll)));
        presid[indx - jskip] -= pflux;
        uresid[indx - jskip] -= uflux;
        vresid[indx - jskip] -= vflux;
        wresid[indx - jskip] -= wflux;
        __syncthreads();
        presid[indx] += pflux;
        uresid[indx] += uflux;
        vresid[indx] += vflux;
        wresid[indx] += wflux;
    }
}

__global__ void computeResidual_z_kernel(float *presid, float *uresid, float *vresid, float *wresid,
                    const float *p, const float *u, const float *v, const float *w,
                    float eta, float nu, float dx, float dy, float dz,
                    int ni, int nj, int nk, int kstart, int iskip, int jskip) {
    const float vcoef = nu / dz;
    const float area = dx * dy;
    // k dimension goes in the +z coordinate direction
    const int kskip = 1;
    int j = blockIdx.x;  // j iteration is assigned to thread blocks
    int k = threadIdx.x; // k iteration is assigned to the threads within each thread block
    // Loop through k faces of the mesh and compute fluxes in z direction
    // Add fluxes to cells that neighbor face
    for (int i = 0; i < ni; i++){
        int offset = kstart + i*iskip + j*jskip;
        const int indx = k + offset;
        // Compute the y direction inviscid flux
        // extract pressures and velocity from the stencil
        float ull = u[indx - 2 * kskip];
        float ul = u[indx - kskip];
        float ur = u[indx];
        float urr = u[indx + kskip];

        float vll = v[indx - 2 * kskip];
        float vl = v[indx - kskip];
        float vr = v[indx];
        float vrr = v[indx + kskip];

        float wll = w[indx - 2 * kskip];
        float wl = w[indx - kskip];
        float wr = w[indx];
        float wrr = w[indx + kskip];

        float pll = p[indx - 2 * kskip];
        float pl = p[indx - kskip];
        float pr = p[indx];
        float prr = p[indx + kskip];
        float pterm = (2. / 3.) * (pl + pr) - (1. / 12.) * (pl + pr + pll + prr);
        // y direction so the flux will be a function of v
        float udotn1 = wl + wr;
        float udotn2 = wl + wrr;
        float udotn3 = wll + wr;
        float pflux = eta*((2./3.)*udotn1 - (1./12.)*(udotn2 + udotn3));
        float uflux = ((1./3.)*(ul + ur)*udotn1 - (1./24.)*((ul + urr)*udotn2 + (ull + ur)*udotn3));
        float vflux = ((1./3.)*(vl + vr)*udotn1 - (1./24.)*((vl + vrr)*udotn2 + (vll + vr)*udotn3));
        float wflux = ((1./3.)*(wl + wr)*udotn1 - (1./24.)*((wl + wrr)*udotn2 + (wll + wr)*udotn3) + pterm);

        // Add in viscous fluxes integrate over face area
        pflux *= area;
        uflux = area * (uflux - vcoef * ((5. / 4.) * (ur - ul) - (1. / 12.) * (urr - ull)));
        vflux = area * (vflux - vcoef * ((5. / 4.) * (vr - vl) - (1. / 12.) * (vrr - vll)));
        wflux = area * (wflux - vcoef * ((5. / 4.) * (wr - wl) - (1. / 12.) * (wrr - wll)));
        presid[indx - kskip] -= pflux;
        uresid[indx - kskip] -= uflux;
        vresid[indx - kskip] -= vflux;
        wresid[indx - kskip] -= wflux;
        __syncthreads();
        presid[indx] += pflux;
        uresid[indx] += uflux;
        vresid[indx] += vflux;
        wresid[indx] += wflux;
    }
}

// Compute the residue which is represent the computed rate of change for the
// pressure and the three components of the velocity vector denoted (u,v,w)
void computeResidual(float *presid, float *uresid, float *vresid, float *wresid,
                    const float *p, const float *u, const float *v, const float *w,
                    float eta, float nu, float dx, float dy, float dz,
                    int ni, int nj, int nk, int kstart, int iskip, int jskip) {
    // i dimension goes in the +x coordinate direction
    computeResidual_x_kernel<<<nj, nk>>>(presid, uresid, vresid, wresid,
                        p, u, v, w, eta, nu, dx, dy, dz, ni, nj, nk, kstart, iskip, jskip);
    computeResidual_y_kernel<<<ni, nk>>>(presid, uresid, vresid, wresid,
                        p, u, v, w, eta, nu, dx, dy, dz, ni, nj, nk, kstart, iskip, jskip);
    computeResidual_z_kernel<<<nj, nk+1>>>(presid, uresid, vresid, wresid,
                        p, u, v, w, eta, nu, dx, dy, dz, ni, nj, nk, kstart, iskip, jskip);
}


// Calculate the stable timestep considering inviscid and viscous terms
__global__ void computeStableTimestep_kernel(float *scratch, const float *u, const float *v, const float *w,
                            float cfl, float eta, float nu, float dx, float dy, float dz,
                            int ni, int nj, int nk, int kstart, int iskip, int jskip){
    float minDt = 1e30;
    int i = blockIdx.x;
    int k = threadIdx.x;
    for (int j = 0; j < nj; ++j){
        int offset = kstart + i*iskip + j*jskip;
        const int indx = k + offset;
        // inviscid timestep
        const float maxu2 = max(u[indx] * u[indx], max(v[indx] * v[indx], w[indx] * w[indx]));
        const float af = sqrt(maxu2 + eta);
        const float maxev = sqrt(maxu2) + af;
        const float sum = maxev * (1. / dx + 1. / dy + 1. / dz);
        minDt = min(minDt, cfl / sum);
        // viscous stable timestep
        const float dist = min(dx, min(dy, dz));
        minDt = min(minDt, 0.2 * cfl * dist * dist / nu);
    }
    // We store the sums over the k iteration into the scratch array
    scratch[i * nk + k] = minDt;
}

__global__ void minKernel(float *mins){
    // each thread within the thread block will used shared memory to
    // compute the min within the thread block using __syncthreads().
    // The final minimization over thread blocks will be performed by the CPU
    int t = threadIdx.x;
    int b = blockIdx.x;
    __shared__ float scratch[1024];
    // There will be 1024 threads used to get the most parallel operations possible.
    // First we copy the value we are combining into shared memory so that threads can work
    // on minimizing the results without touching main memory
    scratch[t] = mins[b * blockDim.x + t]; // load shared memory
    // implement a tree based reduction strategy where the size of scratch is halved each step.
    int nthreads = blockDim.x;
    int offset = nthreads;
    // offset is set to the size of the scratch array
    while (offset > 2) { // sum into shared
        offset >>= 1; // offset = offset / 2
        __syncthreads();
        // check if last update to scratch has been completed by all threads in block
        if (t < offset) // if our thread is writing to memory, write minimum
            scratch[t] = min(scratch[t], scratch[t + offset]);
    }
    __syncthreads();
    // Final step writes result from each thread block into mins array
    if (t == 0)
        mins[b] = min(scratch[0], scratch[1]);
}


// Compute the fluid kinetic energy contained within the simulation domain
// This is part of a 2 part kernel for summing the kinetic enery for the mesh
__global__ void integrateKineticEnergy_kernel(float *scratch,
                                            const float *u, const float *v, const float *w,
                                            float dx, float dy, float dz, int ni, int nj, int nk,
                                            int kstart, int iskip, int jskip) {
    double vol = dx * dy * dz;
    double sum = 0;
    // Threads are allocated to iterations as was done in initial conditions
    int i = blockIdx.x;
    int k = threadIdx.x;
    for (int j = 0; j < nj; ++j){
        int offset = kstart + i * iskip + j * jskip;
        const int indx = k + offset;
        const float udotu = u[indx] * u[indx] + v[indx] * v[indx] + w[indx] * w[indx];
        sum += 0.5 * vol * udotu;
    }
    // We store the sums over the k iteration into the scratch array
    scratch[i * nk + k] = sum;
}

__global__ void sumKernel(float *sum){
    // each thread in a thread block will used shared memory to compute sums
    // within the thread block (using a __syncthreads() call).
    // final summation over thread blocks will be performed by the CPU
    int t = threadIdx.x;
    int b = blockIdx.x;
    __shared__ float scratch[1024];
    scratch[t] = sum[b * blockDim.x + t]; // load shared memory
    // implements a tree based summing strategy where the size of scratch is halved each step
    int nthreads = blockDim.x;
    int offset = nthreads;
    // offset is set to the size of the scratch array
    while (offset > 2) {
        // compute offset to paired number
        offset >>= 1; // right bitshift by 1
        __syncthreads();
        // checks if previous update to scratch has been completed by all threads in block
        if (t < offset) // if our thread is writing to memory ,write sum
            scratch[t] += scratch[t + offset];
    }
    __syncthreads();
    // Final step writes result from each thread block into sum array
    if (t == 0)
        sum[b] = scratch[0] + scratch[1];
}

// Perform a weighted sum of three arrays
// Note, the last weight is used for the input array (no aliasing)
__global__ void weightedSum3_kernel(float *uout, float w1, const float *u1, float w2, const float *u2, float w3,
                int ni, int nj, int nk, int kstart, int iskip, int jskip) {
    int i = blockIdx.x;
    int k = threadIdx.x;
    for(int j = 0; j < nj; ++j) {
        int offset = kstart + i*iskip + j*jskip;
        const int idx = k + offset;
        uout[idx] = w1*u1[idx] + w2*u2[idx] + w3*uout[idx];
    }
}

int parse_arg(char* av) {
    const char* flag_str[11] = { "-n","-ni","-nj","-nk","-L","-nu","-refVel","-stopTime","-cflmax","-outfile","-o" };
    for (int i = 0; i < 11; i++) {
        if (!strcmp(av, flag_str[i]))
            return i;
    }
    return -1;
}

int main(int ac, char *av[]){
    // Default Simulation Parameters
    // Dimensions of the simulation mesh
    int ni = 32;
    int nj = 32;
    int nk = 32;
    // Length of the cube
    float L = 6.28318530718;
    // fluid viscosity
    float nu = 0.000625;
    // Reference velocity used for artificial compressibility apprach
    float refVel = 10;
    // Simulation stopping time
    float stopTime = 20;
    // Coefficient used to compute stable timestep
    float cflmax = 1.9;

    string outfile = "fke.dat";
    // parse command line arguments
    while (ac >= 2 && av[1][0] == '-') {
        int flag = parse_arg(av[1]);
        if ((ac >= 3) && (flag >= 0) && (flag < 11)) {
            switch (flag) {
            case 0:
                ni = atoi(av[2]);
                nj = ni;
                nk = ni;
                break;
            case 1:
                ni = atoi(av[2]); break;
            case 2:
                nj = atoi(av[2]); break;
            case 3:
                nk = atoi(av[2]); break;
            case 4:
                L = atof(av[2]); break;
            case 5:
                nu = atof(av[2]); break;
            case 6:
                refVel = atof(av[2]); break;
            case 7:
                stopTime = atof(av[2]); break;
            case 8:
                cflmax = atof(av[2]); break;
            case 9: // -outfile and -o have the same outcome
            case 10:
                outfile = av[2]; break;
            }
            av += 2;
            ac -= 2;
        }
        else {
            cerr << "unknown command line argument '" << av[1] << "'" << endl;
            av += 1;
            ac -= 1;
            exit(-1);
        }
    }
    // File to save the fluid kinetic energy history
    ofstream ke_file(outfile.c_str(), ios::trunc);

    // Eta is a artificial compressibility parameter to the numerical scheme
    float eta = refVel * refVel;

    // The mesh cell sizes
    float dx = L / ni;
    float dy = L / nj;
    float dz = L / nk;

    struct timeval tval_start, tval_end, tval_elapsed;
    gettimeofday(&tval_start, 0);
    //  Allocate a 3-D mesh with enough space ghost cells two layers thick on each
    //  side of the mesh.
    int allocsize = (ni + 4) * (nj + 4) * (nk + 4);
    cout.precision(5);
    cout << "allocating " << ((allocsize * 4 * 3 * sizeof(float)) >> 10) << " k bytes for fluid computation" << endl;
    // Fluid pressure and velocity

    /* vector<float> p(allocsize);
    vector<float> u(allocsize);
    vector<float> v(allocsize);
    vector<float> w(allocsize);
    // scratch space used to estimate the next timestep values in
    // time integration
    vector<float> pnext(allocsize);
    vector<float> unext(allocsize);
    vector<float> vnext(allocsize);
    vector<float> wnext(allocsize);
    // scratch space to store residual
    vector<float> presid(allocsize);
    vector<float> uresid(allocsize);
    vector<float> vresid(allocsize);
    vector<float> wresid(allocsize);*/
    // Allocate on cuda side
    float *p_cuda = allocate_gpu(allocsize, "p");
    float *u_cuda = allocate_gpu(allocsize, "u");
    float *v_cuda = allocate_gpu(allocsize, "v");
    float *w_cuda = allocate_gpu(allocsize, "w");
    // scratch space used to estimate the next timestep values in
    // time integration
    float *pnext_cuda = allocate_gpu(allocsize, "pnext");
    float *unext_cuda = allocate_gpu(allocsize, "unext");
    float *vnext_cuda = allocate_gpu(allocsize, "vnext");
    float *wnext_cuda = allocate_gpu(allocsize, "wnext");
    // scratch space to store residual
    float *presid_cuda = allocate_gpu(allocsize, "presid");
    float *uresid_cuda = allocate_gpu(allocsize, "uresid");
    float *vresid_cuda = allocate_gpu(allocsize, "vresid");
    float *wresid_cuda = allocate_gpu(allocsize, "wresid");

    float *scratch_cuda = allocate_gpu(ni * nk, "scratch");

    int iskip = (nk + 4) * (nj + 4);
    int jskip = (nk + 4);
    int kstart = 2 * iskip + 2 * jskip + 2;

    // Setup initial conditions
    // no need to copy cuda vectors to the gpu since they're empty here and aren't used in computation
    setInitialConditions_kernel<<<ni, nk>>>(p_cuda, u_cuda, v_cuda, w_cuda,
                                            ni, nj, nk, kstart, iskip, jskip, L);

    // Find initial integrated fluid kinetic energy to monitor solution
    integrateKineticEnergy_kernel<<<ni, nk>>>(scratch_cuda, u_cuda, v_cuda, w_cuda, dx, dy, dz,
                                            ni, nj, nk, kstart, iskip, jskip);
    int ntot = ni*nk;
    int nblocks = ntot >> 10; // ntot/(2^10)
    sumKernel<<<nblocks, 1024>>>(scratch_cuda);
    vector<float> tmp(nblocks);
    copy_gpu_to_cpu(scratch_cuda, &tmp[0], nblocks, "sum");
    float kprev = tmp[0];
    for (int i = 1; i < nblocks; ++i)
        kprev += tmp[i];

    // We use this scaling parameter so we can plot normalized kinetic energy
    float kscale = 1. / kprev;

    // Starting simulation time
    float simTime = 0;
    int iter = 0;

    //float dt = computeStableTimestep(&u[0], &v[0], &w[0], cflmax, eta, nu, dx, dy, dz,
                                    //ni, nj, nk, kstart, iskip, jskip);
    computeStableTimestep_kernel<<<ni, nk>>>(scratch_cuda, u_cuda, v_cuda, w_cuda, cflmax, eta, nu, dx, dy, dz,
                                    ni, nj, nk, kstart, iskip, jskip);
    minKernel<<<nblocks, 1024>>>(scratch_cuda);
    copy_gpu_to_cpu(scratch_cuda, &tmp[0], nblocks, "min");
    float dt = tmp[0];
    for (int i = 1; i < nblocks; ++i)
        dt = min(tmp[i], dt);

    // begin Runge-Kutta 3rd Order Time Integration
    while (simTime < stopTime){
        // copy data to the ghost cells to implement periodic boundary conditions
        copyPeriodic_kernel<<<ni, nk>>>(p_cuda, u_cuda, v_cuda, w_cuda, ni, nj, nk, kstart, iskip, jskip);

        // Zero out the residual function
        // no need to move residuals to gpu first since they're not used in computation
        zeroResidual_kernel<<<ni, nk>>>(presid_cuda, uresid_cuda, vresid_cuda, wresid_cuda,
                                        ni, nj, nk, kstart, iskip, jskip);

        // Compute the residual, these will be used to compute the rates of change
        // of pressure and velocity components
        computeResidual(&presid_cuda[0], &uresid_cuda[0], &vresid_cuda[0], &wresid_cuda[0],
                        &p_cuda[0], &u_cuda[0], &v_cuda[0], &w_cuda[0], eta, nu, dx, dy, dz,
                        ni, nj, nk, kstart, iskip, jskip);

        // First Step of the Runge-Kutta time integration
        // unext = u^n + dt/vol*L(u^n)
        weightedSum3_kernel<<<ni,nk>>>(pnext_cuda, 1.0, p_cuda, dt/(dx*dy*dz), presid_cuda, 0.0,
                                ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(unext_cuda, 1.0, u_cuda, dt/(dx*dy*dz), uresid_cuda, 0.0,
                                ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(vnext_cuda, 1.0, v_cuda, dt/(dx*dy*dz), vresid_cuda, 0.0,
                                ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(wnext_cuda, 1.0, w_cuda, dt/(dx*dy*dz), wresid_cuda, 0.0,
                                ni, nj, nk, kstart, iskip, jskip);

        // Now we are evaluating a residual a second time as part of the
        // third order time integration.  The residual is evaluated using
        // the first estimate of the time integrated solution that is found
        // in next version of the variables computed by the previous
        // wieghtedSum3 calls.

        // Now we are on the second step of the Runge-Kutta time integration
        copyPeriodic_kernel<<<ni, nk>>>(pnext_cuda, unext_cuda, vnext_cuda, wnext_cuda,
                                        ni, nj, nk, kstart, iskip, jskip);

        zeroResidual_kernel<<<ni, nk>>>(presid_cuda, uresid_cuda, vresid_cuda, wresid_cuda,
                    ni, nj, nk, kstart, iskip, jskip);

        computeResidual(&presid_cuda[0], &uresid_cuda[0], &vresid_cuda[0], &wresid_cuda[0],
                        &pnext_cuda[0], &unext_cuda[0], &vnext_cuda[0], &wnext_cuda[0],
                        eta, nu, dx, dy, dz, ni, nj, nk, kstart, iskip, jskip);

        // Second Step of the Runge-Kutta time integration
        // unext = 3/4 u^n + 1/4 u_next + (1/4)*(dt/vol)*L(unext)
        weightedSum3_kernel<<<ni,nk>>>(pnext_cuda, 3./4., p_cuda, dt/(4.*dx*dy*dz), presid_cuda,
                                        1./4., ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(unext_cuda, 3./4., u_cuda, dt/(4.*dx*dy*dz), uresid_cuda,
                                        1./4., ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(vnext_cuda, 3./4., v_cuda, dt/(4.*dx*dy*dz), vresid_cuda,
                                        1./4., ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(wnext_cuda, 3./4., w_cuda, dt/(4.*dx*dy*dz), wresid_cuda,
                                        1./4., ni, nj, nk, kstart, iskip, jskip);

        // Now we are evaluating the final step of the Runge-Kutta time integration
        // so we need to revaluate the residual on the pnext values

        copyPeriodic_kernel<<<ni, nk>>>(pnext_cuda, unext_cuda, vnext_cuda, wnext_cuda,
                                        ni, nj, nk, kstart, iskip, jskip);


        zeroResidual_kernel<<<ni, nk>>>(presid_cuda, uresid_cuda, vresid_cuda, wresid_cuda,
                    ni, nj, nk, kstart, iskip, jskip);

        computeResidual(&presid_cuda[0], &uresid_cuda[0], &vresid_cuda[0], &wresid_cuda[0],
                        &pnext_cuda[0], &unext_cuda[0], &vnext_cuda[0], &wnext_cuda[0],
                        eta, nu, dx, dy, dz, ni, nj, nk, kstart, iskip, jskip);

        // Third Step of the Runge-Kutta time integration
        // u^{n+1} = 1/3 u^n + 2/3 unext + (2/3)*(dt/vol)*L(unext)
        // Note, here we are writing the result into the previous timestep so that we will
        //be ready to proceed to the next iteration when this step is finished.
        weightedSum3_kernel<<<ni,nk>>>(p_cuda, 2./3., pnext_cuda, 2.*dt/(3.*dx*dy*dz), presid_cuda,
                                        1./3., ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(u_cuda, 2./3., unext_cuda, 2.*dt/(3.*dx*dy*dz), uresid_cuda,
                                        1./3., ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(v_cuda, 2./3., vnext_cuda, 2.*dt/(3.*dx*dy*dz), vresid_cuda,
                                        1./3., ni, nj, nk, kstart, iskip, jskip);
        weightedSum3_kernel<<<ni,nk>>>(w_cuda, 2./3., wnext_cuda, 2.*dt/(3.*dx*dy*dz), wresid_cuda,
                                        1./3., ni, nj, nk, kstart, iskip, jskip);

        // Update the simulation time
        simTime += dt;
        iter++;

        // Collect information on the state of kinetic energy in the system
        integrateKineticEnergy_kernel<<<ni, nk>>>(scratch_cuda, u_cuda, v_cuda, w_cuda, dx, dy, dz,
                                            ni, nj, nk, kstart, iskip, jskip);
        sumKernel<<<nblocks, 1024>>>(scratch_cuda);
        vector<float> tmp(nblocks);
        copy_gpu_to_cpu(scratch_cuda, &tmp[0], nblocks, "sum");
        float knext = tmp[0];
        for (int i = 1; i < nblocks; ++i)
            knext += tmp[i];

        // write out the data for post processing analysis
        ke_file << simTime << " " << kscale * knext << " " << -kscale * (knext - kprev) / dt << endl;
        // Every 128 iterations report the state so we can observe progress of
        // the simulation
        if ((iter & 0x7f) == 0)
            cout << "ke: " << simTime << ' ' << kscale * knext << endl;
        // keep track of the change in kinetic energy over timesteps so we can plot
        // the derivative of the kinetic energy with time.
        kprev = knext;
    }

    gettimeofday(&tval_end, 0);
    timersub(&tval_end, &tval_start, &tval_elapsed);
    double milliseconds = tval_elapsed.tv_sec * 1000.0 + tval_elapsed.tv_usec * 0.001;
    cout << "fluid execution time = " << milliseconds << " ms" << endl;

    cout << "time per cell per timestep = " << 1e6 * milliseconds / (double(ni) * double(nj) * double(nk) * double(iter)) << " ns" << endl;
    cout << "finished with iter = " << iter << endl;
    // release gpu memory
    free_gpu(p_cuda);
    free_gpu(u_cuda);
    free_gpu(v_cuda);
    free_gpu(w_cuda);
    // scratch space used to estimate the next timestep values in
    // time integration
    free_gpu(pnext_cuda);
    free_gpu(unext_cuda);
    free_gpu(vnext_cuda);
    free_gpu(wnext_cuda);
    // scratch space to store residual
    free_gpu(presid_cuda);
    free_gpu(uresid_cuda);
    free_gpu(vresid_cuda);
    free_gpu(wresid_cuda);

    free_gpu(scratch_cuda);
    return 0;
}